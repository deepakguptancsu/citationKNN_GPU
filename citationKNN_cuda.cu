#include "hip/hip_runtime.h"
#include<iostream>
#include "dataReader.h"
#include<float.h>
#include<math.h>
#include<numeric>
#include<algorithm>
#include<sys/time.h>

using namespace std;

#define FILENAME "clean2.data"
#define R_VALUE 3
//#define debug

__global__ void calEucledianDist( int *distMatrix, int columnSize, float *euclDistMatrix, int numOfRows)
{
	//int i = blockcounter;
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	int j = blockDim.y*blockIdx.y+threadIdx.y;

	if(i>=numOfRows || j>=numOfRows)
		return;

	int index = i*numOfRows + j;
	int mirrorIndex = j*numOfRows + i;

	float euclideanDist = 0;
	for(int col=0; col<columnSize; col++)
		euclideanDist += pow(distMatrix[i*columnSize + col]-distMatrix[j*columnSize + col], 2); 
	
	euclideanDist = sqrt(euclideanDist);

	euclDistMatrix[index] = euclideanDist;
	euclDistMatrix[mirrorIndex] = euclideanDist;
}


__global__ void calHousdroffDist( int columnSize, int *bucketLocArr, int *bucketSizeArr, float *housdroffDistMatrix, float *euclDistMatrix, int euclDistColSize)
{
	int i = blockIdx.x;
	int j = threadIdx.x;

	#ifdef debug
	printf("Thread: %dx%d starting!\n", i, j);
	#endif

	if(i == j)
	{
		housdroffDistMatrix[i*columnSize + i] = FLT_MAX;
		return;
	}

	if(i > j)
		return;
	
	//calculating housdroff Distance between two bags
	//i represents bucket A while j represents bucket B
	//below for loops are similar to function call dist(A,B)
	//which returns housdroff distance between bucket A and B
	
	float d_A_B = FLT_MIN;
	float d_B_A = FLT_MIN;

	#ifdef debug
	printf("Thread: %dx%d going into bucket loop:\n", i, j);//debug
	#endif

	int bucketA_counter_initialize = bucketLocArr[i];
	int bucketA_counter_end = bucketLocArr[i]+bucketSizeArr[i];

	int bucketB_counter_initialize = bucketLocArr[j];
	int bucketB_counter_end = bucketLocArr[j]+bucketSizeArr[j];

	for(int bucketA_counter = bucketA_counter_initialize; bucketA_counter<bucketA_counter_end; bucketA_counter++)
	{
		float d_ai_B = FLT_MAX;

		for(int bucketB_counter = bucketB_counter_initialize; bucketB_counter<bucketB_counter_end; bucketB_counter++)
		{
			//calculate euclidean distance between rows represented by 
			//bucketA_counter and bucketB_counter
			float euclideanDist = euclDistMatrix[bucketA_counter*euclDistColSize + bucketB_counter];
			d_ai_B = min(d_ai_B, euclideanDist);
		}

		d_A_B = max(d_A_B, d_ai_B);
	}

	#ifdef debug
	printf("Thread: %dx%d going into d_bi_A loop:\n", i, j);//debug
	#endif

	for(int bucketB_counter = bucketB_counter_initialize; bucketB_counter<bucketB_counter_end; bucketB_counter++)
	{
		float d_bi_A = FLT_MAX;
		
		for(int bucketA_counter = bucketA_counter_initialize; bucketA_counter<bucketA_counter_end; bucketA_counter++)
		{
			//calculate euclidean distance between rows represented by 
			//bucketA_counter and bucketB_counter
			float euclideanDist = euclDistMatrix[bucketB_counter*euclDistColSize + bucketA_counter];
			d_bi_A = min(d_bi_A, euclideanDist);
		}

		d_B_A = max(d_B_A, d_bi_A);
	}

	float H_A_B = max(d_A_B, d_B_A);
	housdroffDistMatrix[i*columnSize + j] = H_A_B;
	housdroffDistMatrix[j*columnSize + i] = H_A_B;

	#ifdef debug
	printf("Thread: %d x %d done!\n", i, j);//debug
	#endif
}



//R is number of references
//C is number of citors
__global__ void calculateVoterList(int queryBag, int numOfBuckets, float *housdroffDistMatrix, int *classLabel, int *voterList, int R, int C)
{
	int voterListVal = 0;
	int closestNeighbourIndices[10];
	float maxVal = FLT_MIN;
	int maxClosestNeighbourIdx;
	int bucketCounter = threadIdx.x;
	int X;

	if(bucketCounter == queryBag)
	{
		X = R;
	}
	else
	{
		X = C;
	}

	for(int i=0; i<X; i++)
	{
		closestNeighbourIndices[i] = i;
		if(maxVal < housdroffDistMatrix[bucketCounter*numOfBuckets + i])
		{
			maxVal = housdroffDistMatrix[bucketCounter*numOfBuckets + i];
			maxClosestNeighbourIdx = i;
		}
	}

	for(int i=X; i<numOfBuckets; i++)
	{
		if(maxVal > housdroffDistMatrix[bucketCounter*numOfBuckets + i])
		{
			closestNeighbourIndices[maxClosestNeighbourIdx] = i;
			float tempMax = FLT_MIN;

			for(int j=0; j<X; j++)
			{
				if(tempMax < housdroffDistMatrix[bucketCounter*numOfBuckets + closestNeighbourIndices[j]])
				{
					tempMax = housdroffDistMatrix[bucketCounter*numOfBuckets + closestNeighbourIndices[j]];
					maxClosestNeighbourIdx = j;
				}
			}

			maxVal = tempMax;
		}
	}

	if(bucketCounter == queryBag)
	{
		for(int i=0; i<X; i++)
		{
			if(classLabel[closestNeighbourIndices[i]] == 1)
				voterListVal++;
			else
				voterListVal--;
		}
	}
	else
	{
		for(int i=0; i<X; i++)
		{
			if(queryBag == closestNeighbourIndices[i])
			{
				if(classLabel[closestNeighbourIndices[i]] == 1)
					voterListVal++;
				else
					voterListVal--;

				break;
			}
		}
	}

	voterList[bucketCounter] = voterListVal;
}

int main()
{
	testDataSet obj;
	obj.readData(FILENAME);
	
	int R = R_VALUE;
	int C = R + 2;

	float correctPrediction = 0;
	int numOfBuckets = obj.bucketSizeArr.size();

	#ifdef debug
	cout<<"total rows: "<<(obj.distMatrix.size())/obj.columnSize<<endl;
	cout<<"total columns: "<<obj.columnSize<<endl;
	#endif

	float *housdroffDistMatrix = (float *) malloc(sizeof(float) * numOfBuckets * numOfBuckets);

	#ifdef debug	
	cout<<"obj.distMatrix.size() "<<obj.distMatrix.size()<<endl;
	#endif

	int *d_distMatrix;
	if(!hipMalloc(&d_distMatrix, sizeof(int) * obj.distMatrix.size()) == hipSuccess)
		cout<<"error in allocating distMatrix\n";
	if(!hipMemcpy(d_distMatrix, obj.distMatrix.data(), sizeof(int) * obj.distMatrix.size(), hipMemcpyHostToDevice) == hipSuccess)
		cout<<"error in copying distMatrix\n";

	int numOfRows = (obj.distMatrix.size())/obj.columnSize;
	cout<<"numRows"<<numOfRows<<endl;
	
	float *d_euclDistMatrix;
	if(!hipMalloc(&d_euclDistMatrix, sizeof(float) * numOfRows * numOfRows) == hipSuccess)
		cout<<"error in allocating d_euclDistMatrix\n";

	//for(int blockcounter = 0; blockcounter<numOfRows; blockcounter++)
	//	calEucledianDist<<<1,numOfRows>>>(d_distMatrix, obj.columnSize, d_euclDistMatrix, blockcounter);
	//calEucledianDist<<<numOfRows,numOfRows>>>(d_distMatrix, obj.columnSize, d_euclDistMatrix);

	dim3 threadDimension(32,32);
	dim3 blockDimension(numOfRows/32 + 1, numOfRows/32 + 1);
	calEucledianDist<<<blockDimension, threadDimension>>>(d_distMatrix, obj.columnSize, d_euclDistMatrix, numOfRows);

	hipDeviceSynchronize();

	hipFree(d_distMatrix);

#if 0
	//debug code start
	float *h_euclDistMatrix = (float *) malloc(sizeof(float) * numOfRows * numOfRows);
	hipMemcpy(h_euclDistMatrix, d_euclDistMatrix, sizeof(float) * numOfRows * numOfRows, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for(int i=0; i<numOfRows; i++)
	{
		for(int j=0; j<numOfRows; j++)
			printf("%f ", h_euclDistMatrix[i*numOfRows+j]);

		printf("\n");
	}

	free(h_euclDistMatrix);
	//debug code end
#endif

	int *d_bucketLocArr;
	if(!hipMalloc(&d_bucketLocArr, sizeof(int) * obj.bucketLocArr.size())  == hipSuccess)
		cout<<"error in allocating d_bucketLocArr\n";
	if(!hipMemcpy(d_bucketLocArr, obj.bucketLocArr.data(), sizeof(int) * obj.bucketLocArr.size(), hipMemcpyHostToDevice) == hipSuccess)
		cout<<"error in copying d_bucketLocArr\n";
	
	int *d_bucketSizeArr;
	if(!hipMalloc(&d_bucketSizeArr, sizeof(int) * obj.bucketSizeArr.size()) == hipSuccess)
		cout<<"error in allocating d_bucketSizeArr\n";
	if(!hipMemcpy(d_bucketSizeArr, obj.bucketSizeArr.data(), sizeof(int) * obj.bucketSizeArr.size(), hipMemcpyHostToDevice) == hipSuccess)
		cout<<"error in copying d_bucketSizeArr\n";

	float *d_housdroffDistMatrix;
	if(!hipMalloc(&d_housdroffDistMatrix, sizeof(float) * numOfBuckets * numOfBuckets) == hipSuccess)
		cout<<"error in allocating d_housdroffDistMatrix\n";

	cout<<"malloc and hipMemcpy done\n";

	calHousdroffDist<<<numOfBuckets,numOfBuckets>>>(obj.columnSize, d_bucketLocArr, d_bucketSizeArr, d_housdroffDistMatrix, d_euclDistMatrix, numOfRows);
	hipDeviceSynchronize();
	
	cout<<"housdroff distance calculation complete\n";

#if 0
	//debug code start
	cout<<"\n\nprinting housdroff matrix\n\n";
	float *h_housdroffDistMatrix = (float *) malloc(sizeof(float) * numOfBuckets * numOfBuckets);
	hipMemcpy(h_housdroffDistMatrix, d_housdroffDistMatrix, sizeof(float) * numOfBuckets * numOfBuckets, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for(int i=0; i<numOfBuckets; i++)
	{
		for(int j=0; j<numOfBuckets; j++)
			printf("%f ", h_housdroffDistMatrix[i*numOfBuckets+j]);

		printf("\n");
	}

	free(h_housdroffDistMatrix);
	//debug code end
#endif

	hipFree(d_bucketLocArr);
	hipFree(d_bucketSizeArr);
	hipFree(d_euclDistMatrix);

	int *d_classLabel;
	if(!hipMalloc(&d_classLabel, sizeof(int) * obj.classLabel.size()) == hipSuccess)
		cout<<"error in allocating d_classLabel\n";
	if(!hipMemcpy(d_classLabel, obj.classLabel.data(), sizeof(int) * obj.classLabel.size(), hipMemcpyHostToDevice) == hipSuccess)
		cout<<"error in copying d_classLabel\n";

	int *d_voterList;
	if(!hipMalloc(&d_voterList, sizeof(int) * numOfBuckets) == hipSuccess)
		cout<<"error in allocating d_voterList\n";

	int *h_voterList = (int*) malloc(sizeof(int) * numOfBuckets);

	for(int queryBag=0; queryBag<numOfBuckets; queryBag++)
	{
		hipMemset(d_voterList, 0, sizeof(int) * numOfBuckets);
		int classValue = 0;

		calculateVoterList<<<1,numOfBuckets>>>(queryBag, numOfBuckets, d_housdroffDistMatrix, d_classLabel, d_voterList, R, C);
		hipDeviceSynchronize();
		
		hipMemcpy(h_voterList, d_voterList, sizeof(int) * numOfBuckets, hipMemcpyDeviceToHost);
		
		for(int j=0; j<numOfBuckets; j++)
			classValue += h_voterList[j];
		
		if(classValue > 0)
			classValue = 1;
		else
			classValue = 0;

		if(classValue == obj.classLabel[queryBag])
			correctPrediction++;
	}

	cout<<"correctPrediction = "<<correctPrediction<<endl;
	cout<<"dataSize = "<<numOfBuckets<<endl;
	float accuracy = (correctPrediction * 100.0)/(float) numOfBuckets;
	cout<<"accurary is "<<accuracy<<endl;

	hipFree(d_housdroffDistMatrix);
	hipFree(d_classLabel);
	hipFree(d_voterList);

	free(h_voterList);
	free(housdroffDistMatrix);

	return 0;
}